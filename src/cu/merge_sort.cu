#include <libgpu/cuda/cu/opencl_translator.cu>

#include "../cl/merge_sort.cl"

void cuda_merge_sort(const gpu::WorkSize &workSize, hipStream_t stream,
					 const float* as, float* bs, unsigned int n, unsigned int sorted_chunks_size) {
	merge_sort<<<workSize.cuGridSize(), workSize.cuBlockSize()>>>(as, bs, n, sorted_chunks_size);
	CUDA_CHECK_KERNEL(stream);
}
